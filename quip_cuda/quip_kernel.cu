#include "hip/hip_runtime.h"
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <mma.h>

#include <ATen/ATen.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/DeviceGuard.h>
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>


template <typename U, typename V>
constexpr __host__ __device__ auto divDown(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral<U>::value && std::is_integral<V>::value, "");
  return (a / b);
}

template <typename U, typename V>
constexpr __host__ __device__ auto divUp(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral<U>::value && std::is_integral<V>::value, "");
  // Overflow safe variant of (a + b - 1) / b
  const uint64_t blocks = a / b + (a % b != 0);
  return blocks;
}

template <typename U, typename V>
constexpr __host__ __device__ auto roundDown(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral<U>::value && std::is_integral<V>::value, "");
  return divDown(a, b) * b;
}

template <typename U, typename V>
constexpr __host__ __device__ auto roundUp(U a, V b) -> decltype(a + b) {
  static_assert(std::is_integral<U>::value && std::is_integral<V>::value, "");
  return divUp(a, b) * b;
}

constexpr int32_t kWarpSize = 32;
constexpr int32_t KTilesPerWarp = 8;
constexpr int32_t kMTileSize = 16;
constexpr int32_t kNTileSize = 8;
constexpr int32_t kKTileSize = 16;

struct __align__(16) f16x2x4_u32 {
  uint32_t vals[4];
};
struct __align__(16) f16x2x2_u32 {
  uint32_t vals[2];
};

struct ALayout_RM {
template <int KTilesToLoad>
static __device__ void load(
    const half* A,
    int32_t m,
    int32_t k,
    int32_t mTiles,
    int32_t mTile,
    int32_t kTiles,
    int32_t kTileStart,
    int32_t laneId,
    f16x2x4_u32 out[KTilesToLoad]) {
  const auto mLane = mTile * kMTileSize + (laneId / 4);
  const auto kLane = kTileStart * kKTileSize + (laneId % 4) * 4;

  // access
  // [mTile * kMTileSize + (laneId / 4)]
  // [kTileStart * kKTileSize + (laneId % 4) * 2]
  auto aPtr = A + mLane * k + kLane;

  auto aPtrPlus8Rows = aPtr + 8 * k;

  bool m0InBounds = mLane < m;
  bool m1InBounds = (mLane + 8) < m;

#pragma unroll
  for (int i = 0; i < KTilesToLoad; ++i) {
    out[i].vals[0] = m0InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtr  + i * kKTileSize)
          : uint32_t(0);
    out[i].vals[1] = m1InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtrPlus8Rows  + i * kKTileSize)
          : uint32_t(0);

    out[i].vals[2] = m0InBounds
          ? *reinterpret_cast<const uint32_t*>(aPtr  + i * kKTileSize + 2)
          : uint32_t(0);
    out[i].vals[3] = m1InBounds ? *reinterpret_cast<const uint32_t*>(
                                        aPtrPlus8Rows  + i * kKTileSize + 2)
                                  : uint32_t(0);
  }
}

static __device__ void store(
    half* C,
    int32_t m,
    int32_t n,
    int32_t mOutTiles,
    int32_t mTile,
    int32_t nOutTiles,
    int32_t nTile,
    int32_t laneId,
    const float4& out) {

  // sum.x / sum.y are written at
  // [laneId / 4], [(laneId % 4) * 2, (laneId % 4) * 2 + 1]
  // sum.z / sum.w are written at
  // [8 + (laneId / 4)], [(laneId % 4) * 2, (laneId % 4) * 2 + 1]
  // i.e., same columns, different row.
  const int outRow = mTile * kMTileSize + (laneId / 4);
  const int outCol = nTile * kNTileSize + (laneId % 4) * 2;

  // Pointer where sum.x / sum.y is written
  auto cPtr = C + outRow * n + outCol;

  auto v01 = __float22half2_rn(float2{out.x, out.y});
  auto v23 = __float22half2_rn(float2{out.z, out.w});

  if (outRow < m) {
    *reinterpret_cast<half2*>(cPtr) = v01;
  }

  // sum.z, sum.w at +8 rows from cPtr
  if (outRow + 8 < m) {
    *reinterpret_cast<half2*>(cPtr + 8 * n) = v23;
  }
}
};

struct BLayout_D4 {

template <int KTilesPerIteration>
static __device__ void load(
    const void* __restrict__ B,
    const uint64_t* __restrict__ CB,
    int32_t n,
    int32_t k,
    int32_t nTiles,
    int32_t nTile,
    int32_t kTiles,
    int32_t kTileStart,
    int32_t laneId,
    f16x2x2_u32 b[KTilesPerIteration]) {
  auto Bptr = reinterpret_cast<const uint8_t*>(B);
  #pragma unroll
  for (int i = 0; i < KTilesPerIteration; ++i) {
       const int row = nTile * kNTileSize + laneId / 4;
       const int col = (kTileStart + i) * kKTileSize / 4 + laneId % 4;
       *(reinterpret_cast<uint64_t*>(b[i].vals)) = CB[Bptr[row * k/4 + col]];
  }
}
};


struct BLayout_E8 {

__device__ static inline uint64_t decode8weights(
    uint16_t weight_compressed,
    const int64_t *__restrict__ codebook_abs
) {

    uint32_t bit_shift = (weight_compressed & 1)^1;
    uint8_t bits_sign = (weight_compressed >> 1) & ((1 << 7) - 1);
    uint8_t bits_abs = (weight_compressed >> 8) & ((1 << 9) - 1);

    int64_t packed_ = codebook_abs[bits_abs];
    uint32_t packed[2];
    memcpy(packed, &packed_, sizeof(packed));

    // TODO: optimize this by redefining the bit pattern
    uint32_t parity = __popc(packed[0] & 0x04040404) ^ __popc(packed[1]&0x04040404);
    uint8_t sign_vec = bits_sign | ((__popc(bits_sign) ^ parity) << 7);
    uint32_t decoded_sign[2];
    decoded_sign[0] = sign_vec * 0x08040201ll;
    decoded_sign[1] = sign_vec * 0x80402010ll;
    decoded_sign[0] &= 0x80808080;
    decoded_sign[1] &= 0x80808080;
    decoded_sign[0] >>= 7;
    decoded_sign[1] >>= 7;
    decoded_sign[0] *= 255 - 3;
    decoded_sign[1] *= 255 - 3;
    packed[0] ^= decoded_sign[0];
    packed[1] ^= decoded_sign[1];
    packed[0] |= 0x01010101;
    packed[1] |= 0x01010101;
    packed[0] -= bit_shift * 0x02020202;
    packed[1] -= bit_shift * 0x02020202;

    memcpy(&packed_, packed, sizeof(packed));

    return packed_;
};

template <int KTilesPerIteration>
static __device__ void load(
    const void* __restrict__ B,
    const uint64_t* __restrict__ CB,
    int32_t n,
    int32_t k,
    int32_t nTiles,
    int32_t nTile,
    int32_t kTiles,
    int32_t kTileStart,
    int32_t laneId,
    f16x2x2_u32 b[KTilesPerIteration]) {
  auto Bptr = (const uint16_t*) B;
  #pragma unroll
  for (int i = 0; i < KTilesPerIteration; ++i) {
       // The kernel here is not optimized. Half of the data read
       // and dequant calculate is wasted.
       const int row = nTile * kNTileSize + laneId / 4;
       const int col = (kTileStart + i) * kKTileSize / 8 + laneId % 4 / 2;
       uint64_t decoded = decode8weights(Bptr[row * k/8 + col], (const int64_t*)CB);
       half2 unpacked[2][2];
       uint64_t lower_half = decoded & 0x00ff00ff00ff00ff;
       lower_half = (lower_half ^ 0x6480648064806480);
       memcpy(unpacked[0], &lower_half, sizeof(uint64_t));
       uint64_t upper_half = (decoded & 0xff00ff00ff00ff00) >> 8;
       upper_half = (upper_half ^ 0x6480648064806480);
       memcpy(unpacked[1], &upper_half, sizeof(uint64_t));

       const half adjust_ = __float2half_rn(-288.0f);
       const half factor_ = __float2half(0.25f);
       const half2 adjust = __halves2half2(adjust_, adjust_);
       const half2 factor = __halves2half2(factor_, factor_);
       unpacked[0][0] = __hfma2(unpacked[0][0], factor, adjust);
       unpacked[0][1] = __hfma2(unpacked[0][1], factor, adjust);
       unpacked[1][0] = __hfma2(unpacked[1][0], factor, adjust);
       unpacked[1][1] = __hfma2(unpacked[1][1], factor, adjust);

       *((half*)(b[i].vals)) = unpacked[0][laneId & 1].x;
       *((half*)(b[i].vals) + 1) = unpacked[1][laneId & 1].x;
       *((half*)(b[i].vals) + 2) = unpacked[0][laneId & 1].y;
       *((half*)(b[i].vals) + 3) = unpacked[1][laneId & 1].y;
  }
}
};


template <
    typename ALayout,
    typename BLayout,
    typename CLayout,
    int Warps,
    int KTilesPerIteration>
__global__
__launch_bounds__(256) void tinygemm_m16n8k16_chunk_kernel(
    // Data for the A matrix, loaded as per ALayout
    const half* __restrict__ A,
    const void* __restrict__ B,
    const uint64_t* __restrict__ CB,

    // Output data for the C matrix, stored as per CLayout
    half* __restrict__ C,

    // The size of the matrix multiplication
    int32_t m,
    int32_t n,
    int32_t k,

    // The size of the matrix multiplication, in multiples of our TC tile size
    int32_t mTiles,
    int32_t nTiles,
    int32_t kTiles) {

  __shared__ uint64_t CB_[256];
  CB_[threadIdx.x + threadIdx.y * 32] = CB[threadIdx.x + threadIdx.y * 32];
  __syncthreads();

  auto warpId = threadIdx.y;
  auto laneId = threadIdx.x;

  int32_t mTile = blockIdx.z;
  int32_t nTile = blockIdx.y;

  float4 c{0.0f, 0.0f, 0.0f, 0.0f};

 // First, handle whole multiples of KTilesPerIteration
  auto kTilesLimit = roundDown(kTiles, KTilesPerIteration);

  // Each warp handles a set of KTilesPerIteration under the above limit
  for (int32_t kTileBase = warpId * KTilesPerIteration; kTileBase < kTilesLimit; kTileBase += Warps * KTilesPerIteration) {
    //
    // Load data from A
    //
    f16x2x4_u32 a[KTilesPerIteration];
    ALayout::template load<KTilesPerIteration>(
        A, m, k, mTiles, mTile, kTiles, kTileBase, laneId, a);

    //
    // Load data from B and de-quantize as needed
    //
    f16x2x2_u32 b[KTilesPerIteration];
    BLayout::template load<KTilesPerIteration>(
        B, CB_, n, k, nTiles, nTile, kTiles, kTileBase, laneId, b);

    // Now, perform the matrix multiplication
    //
    #pragma unroll
    for (int i = 0; i < KTilesPerIteration / 2; ++i) {
      float4 cTmp[2];

      #pragma unroll
      for (int k = 0; k < 2; ++k) {
        cTmp[k] = float4{0.0f, 0.0f, 0.0f, 0.0f};
      }

      #pragma unroll
      for (int k = 0; k < 2; ++k) {
        asm volatile(
              "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
              "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};"
              : "=f"(cTmp[k].x),
                "=f"(cTmp[k].y),
                "=f"(cTmp[k].z),
                "=f"(cTmp[k].w)
              : "r"(a[i * 2 + k].vals[0]),
                "r"(a[i * 2 + k].vals[1]),
                "r"(a[i * 2 + k].vals[2]),
                "r"(a[i * 2 + k].vals[3]),
                "r"(b[i * 2 + k].vals[0]),
                "r"(b[i * 2 + k].vals[1]),
                "f"(cTmp[k].x),
                "f"(cTmp[k].y),
                "f"(cTmp[k].z),
                "f"(cTmp[k].w));
      }
      #pragma unroll
      for (int k = 0; k < 2; ++k) {
        c.x += cTmp[k].x;
        c.y += cTmp[k].y;
        c.z += cTmp[k].z;
        c.w += cTmp[k].w;
      }
    }

  } // for all tiles under kTilesLimit


  auto kTileBaseRemaining = kTilesLimit + warpId;

  // If we have any remainder k-tiles, some warps will handle them, processing
  // kInnerKTiles k-tiles at a time
  if (kTileBaseRemaining < kTiles) {
    f16x2x4_u32 a;
    ALayout::template load<1>(
        A, m, k, mTiles, mTile, kTiles, kTileBaseRemaining, laneId, &a);

    f16x2x2_u32 b;
    BLayout::template load<1>(
        B, CB, n, k, nTiles, nTile, kTiles, kTileBaseRemaining, laneId, &b);

    asm volatile(
              "mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
              "{%0,%1,%2,%3}, {%4,%5,%6,%7}, {%8,%9}, {%10,%11,%12,%13};"
              : "=f"(c.x),
                "=f"(c.y),
                "=f"(c.z),
                "=f"(c.w)
              : "r"(a.vals[0]),
                "r"(a.vals[1]),
                "r"(a.vals[2]),
                "r"(a.vals[3]),
                "r"(b.vals[0]),
                "r"(b.vals[1]),
                "f"(c.x),
                "f"(c.y),
                "f"(c.z),
                "f"(c.w));
  }
  // Reduce independent k-tiles (same m/n) across warps
  __shared__ float4 smem_sum[Warps][kWarpSize];

  smem_sum[warpId][laneId] = c;

  __syncthreads();

  if (warpId == 0) {
    float4 sum_f32{0.0f, 0.0f, 0.0f, 0.0f};

    // Reduce across the block in the first warp
    for (int i = 0; i < Warps; ++i) {
      float4 v = smem_sum[i][laneId];
      sum_f32.x += v.x;
      sum_f32.y += v.y;
      sum_f32.z += v.z;
      sum_f32.w += v.w;
    }

    // Write the reduced result (in the first warp) into the output
    CLayout::store(
        C,
        m,
        n,
        mTiles,
        mTile,
        // n for C output becomes k for A input, so for m16n8k16,
        // we need to halve the tiles
        nTiles / 2,
        nTile,
        laneId,
        sum_f32);
  }
}

at::Tensor d4_mm_cuda(
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& CB) {
  c10::cuda::CUDAGuard g(A.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  constexpr int Warps = 8;

  // row major layout
  auto m = A.size(0);
  auto mTiles = divUp(m, kMTileSize);

  // tensor core layout
  auto n = B.size(0);
  auto nTiles = divUp(n, kNTileSize);

  // row major layout
  auto k = A.size(1);
  auto kTiles = divUp(k, kKTileSize);

  // Output is a standard row-major matrix
  auto C_final = at::empty(
      {m, n}, at::TensorOptions().dtype(A.dtype()).device(A.device()));

  auto grid = dim3(1, nTiles, mTiles);
  auto block = dim3(kWarpSize, Warps);
  auto kernel = tinygemm_m16n8k16_chunk_kernel<ALayout_RM, BLayout_D4, ALayout_RM, 8, 8>;

  kernel<<<grid, block, 0, stream>>>(
      (const half*)A.data_ptr(),
      (const void*)B.data_ptr(),
      (const uint64_t*)CB.data_ptr(),
      (half*)C_final.data_ptr(),
      m,
      n,
      k,
      mTiles,
      nTiles,
      kTiles);

  return C_final;
}

at::Tensor e8_mm_cuda(
    const at::Tensor& A,
    const at::Tensor& B,
    const at::Tensor& CB) {
  c10::cuda::CUDAGuard g(A.device());
  auto stream = at::cuda::getCurrentCUDAStream();

  constexpr int Warps = 8;

  // row major layout
  auto m = A.size(0);
  auto mTiles = divUp(m, kMTileSize);

  // tensor core layout
  auto n = B.size(0);
  auto nTiles = divUp(n, kNTileSize);

  // row major layout
  auto k = A.size(1);
  auto kTiles = divUp(k, kKTileSize);

  // Output is a standard row-major matrix
  auto C_final = at::empty(
      {m, n}, at::TensorOptions().dtype(A.dtype()).device(A.device()));

  auto grid = dim3(1, nTiles, mTiles);
  auto block = dim3(kWarpSize, Warps);
  auto kernel = tinygemm_m16n8k16_chunk_kernel<ALayout_RM, BLayout_E8, ALayout_RM, 8, 8>;
  kernel<<<grid, block, 0, stream>>>(
      (const half*)A.data_ptr(),
      (const void*)B.data_ptr(),
      (const uint64_t*)CB.data_ptr(),
      (half*)C_final.data_ptr(),
      m,
      n,
      k,
      mTiles,
      nTiles,
      kTiles);

  return C_final;
}

#define DECOMPRESS_D4_BLOCK_SIZE 256

__global__ void cuda_decompress_d4_origorder_kernel(
    const uint8_t* __restrict__ YIs,	  // m x (n/4)
    const c10::Half* __restrict__ CB,           // 256 x 4
    c10::Half* __restrict__ Y             // m x n
) {
  const long i = threadIdx.x + DECOMPRESS_D4_BLOCK_SIZE * blockIdx.x;

  for(long r = 0; r < 4; r++) {
    uint8_t yidx = ((uint8_t*)YIs)[i*4 + r];
    ((uint64_t*)Y)[i*4 + r] = ((uint64_t*)CB)[yidx & 255];
  }
}


void decompress_d4_origorder(
    torch::Tensor YIs,      // m x (n/4)
    torch::Tensor CB,       // 256 x 4
    torch::Tensor Y         // m x n
) {
  size_t m = Y.sizes()[0];
  size_t n = Y.sizes()[1];

  assert(YIs.is_contiguous());
  assert(CB.is_contiguous());
  assert(Y.is_contiguous());

  assert(YIs.sizes()[0] == m);
  assert(YIs.sizes()[1] * 4 == n);
  assert(CB.sizes()[0] == 256);
  assert(CB.sizes()[1] == 4);

  const dim3 threads(DECOMPRESS_D4_BLOCK_SIZE);
  const dim3 blocks(m*n/(16*DECOMPRESS_D4_BLOCK_SIZE));
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  cuda_decompress_d4_origorder_kernel<<<blocks, threads, 0, stream>>>(
    YIs.data_ptr<uint8_t>(),
    CB.data_ptr<c10::Half>(),
    Y.data_ptr<c10::Half>()
  );
}

#define DECOMPRESS_E8P_BLOCK_SIZE 256
#define FLIP_MASK 9223512776490647552LLU // (1 << 63) + (1 << 47) + (1 << 31) + (1 << 15)

__global__ void cuda_decompress_e8p_origorder_kernel(
    const int16_t* __restrict__ YIs,	  // m x (n/8)
    const c10::Half* __restrict__ CB, // 256 x 8
    const bool* __restrict__ CB_even_flips,
    c10::Half* __restrict__ Y             // m x n
) {
  const long i = threadIdx.x + DECOMPRESS_E8P_BLOCK_SIZE * blockIdx.x;

  uint16_t yidx = ((uint16_t*)YIs)[i] - 32768;
  uint16_t abs_idx = (yidx & 65280) >> 8;
  uint16_t flips = (yidx & 254) >> 1;
  flips |= (((__popc(flips) & 1) == CB_even_flips[abs_idx]) << 7);

  ((uint64_t*)Y)[i*2] = ((uint64_t*)CB)[abs_idx*2];
  uint64_t l4flips = (uint64_t)(flips >> 4);
  l4flips |= (l4flips << 34);
  l4flips |= (l4flips << 17);
  l4flips = (l4flips << 12);
  l4flips &= FLIP_MASK;
  ((uint64_t*)Y)[i*2] |= l4flips;

  ((uint64_t*)Y)[i*2 + 1] = ((uint64_t*)CB)[abs_idx*2 + 1];
  uint64_t r4flips = (uint64_t)(flips & 15);
  r4flips |= (r4flips << 34);
  r4flips |= (r4flips << 17);
  r4flips = (r4flips << 12);
  r4flips &= FLIP_MASK;
  ((uint64_t*)Y)[i*2 + 1] |= r4flips;

  __half2 const shift = (yidx & 1 ? __half2half2((c10::Half)0.25) : __half2half2((c10::Half)-0.25));
# pragma unroll 4
  for(long k = 0; k < 4; k++){
    ((__half2*)Y)[i*4 + k] = __hadd2(((__half2*)Y)[i*4 + k], shift);
  }
}


void decompress_e8p_origorder(
    torch::Tensor YIs,      // m x (n/8)
    torch::Tensor CB,       // 256 x 8
    torch::Tensor CB_even_flips, // 256
    torch::Tensor &Y         // m x n
) {
  size_t m = Y.sizes()[0];
  size_t n = Y.sizes()[1];

  assert(YIs.is_contiguous());
  assert(CB.is_contiguous());
  assert(CB_even_flips.is_contiguous());
  assert(Y.is_contiguous());

  assert(YIs.sizes()[0] == m);
  assert(YIs.sizes()[1] * 8 == n);
  assert(CB.sizes()[0] == 256);
  assert(CB.sizes()[1] == 8);
  assert(CB_even_flips.sizes()[0] == 256);

  const dim3 threads(DECOMPRESS_E8P_BLOCK_SIZE);
  const dim3 blocks(m*n/(8*DECOMPRESS_E8P_BLOCK_SIZE));
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  cuda_decompress_e8p_origorder_kernel<<<blocks, threads, 0, stream>>>(
    YIs.data_ptr<int16_t>(),
    CB.data_ptr<c10::Half>(),
    CB_even_flips.data_ptr<bool>(),
    Y.data_ptr<c10::Half>()
  );
}
