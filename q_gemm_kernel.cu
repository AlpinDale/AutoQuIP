#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <torch/python.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>


#if (defined(__CUDA_ARCH__) && __CUDA_ARCH__ < 700) || defined(USE_ROCM)
// adapted from https://github.com/torch/cutorch/blob/master/lib/THC/THCAtomics.cuh

__device__ __forceinline__ void atomicAdd(c10::Half* address, c10::Half val) {
    unsigned int *address_as_ui = reinterpret_cast<unsigned int *>(reinterpret_cast<char *>(address) - (reinterpret_cast<size_t>(address) & 2));
    unsigned int old = *address_as_ui;
    unsigned int assumed;

    do {
        assumed = old;
        unsigned short hsum = reinterpret_cast<size_t>(address) & 2 ? (old >> 16) : (old & 0xffff);
        hsum += val;
        old = reinterpret_cast<size_t>(address) & 2
                 ? (old & 0xffff) | (hsum << 16)
                 : (old & 0xffff0000) | hsum;
        old = atomicCAS(address_as_ui, assumed, old);

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);
}
__device__ __forceinline__ void atomicAdd(__half* address, c10::Half val) {
    unsigned int * address_as_ui = (unsigned int *) ((char *)address - ((size_t)address & 2));
    unsigned int old = *address_as_ui;
    unsigned int assumed;

    do {
        assumed = old;
        __half_raw hsum;
        hsum.x = (size_t)address & 2 ? (old >> 16) : (old & 0xffff);
        half tmpres = __hadd(hsum, val);
        hsum = __half_raw(tmpres);
        old = (size_t)address & 2 ? (old & 0xffff) | (hsum.x << 16) : (old & 0xffff0000) | hsum.x;
        old = atomicCAS(address_as_ui, assumed, old);
    } while (assumed != old);
}
#endif

__global__ void VecQuant2MatMulKernel(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    int batch,
    int vec_height,
    int height,
    int width
);

__global__ void VecQuant3MatMulKernel(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    int batch,
    int vec_height,
    int height,
    int width
);

__global__ void VecQuant4MatMulKernel(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    int batch,
    int vec_height,
    int height,
    int width
);

const int BLOCKWIDTH  = 256;
const int BLOCKHEIGHT2 =  16;
const int BLOCKHEIGHT3 =  24;
const int BLOCKHEIGHT4 =  32;

__device__ inline unsigned int as_unsigned(int i) {
  return *reinterpret_cast<unsigned int*>(&i);
}

__device__ inline int as_int(int i) {
  return *reinterpret_cast<int*>(&i);
}

void vecquant2matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1) / 2;
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT2 - 1) / BLOCKHEIGHT2,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant2MatMulKernel<<<blocks, threads>>>(
    (half2*) vec.data_ptr(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    batch, vec_height, height, width
  );
}

__global__ void VecQuant2MatMulKernel(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
	  int batch,
	  int vec_height,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKHEIGHT2 * 8;
  int b = blockIdx.z;
  int h = BLOCKHEIGHT2 * blockIdx.x;
  int hr = (min(height, h + BLOCKHEIGHT2) - h) * 8;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];
  if (threadIdx.x < hr)
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * blockwidth2 + threadIdx.x];

  __shared__ half2 deq2[16][16];
  int val = threadIdx.x / 16;
  int off = threadIdx.x % 16;
  for (; val < 16; val += BLOCKWIDTH / 16) {
    deq2[val][off] = __halves2half2(
       __int2half_rn(val & 0x3), __int2half_rn(val >> 2)
    );
  }
  if (w >= width) return;

  int i = width * h + w;
  int k = 0;

  float res = 0;
  half2 res2;
  half2 scale = __float2half2_rn(scales[w] / 3.0f * 2.0f);
  half2 zero = __float2half2_rn(-scales[w]);

  unsigned int tmp;

  __syncthreads();

  while (k < hr) {
    std::memset(&res2, 0, sizeof(half2));
    tmp = as_unsigned(mat[i]);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  0) & 0xf][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  4) & 0xf][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  8) & 0xf][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 12) & 0xf][off], scale, zero), blockvec[k + 3], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 16) & 0xf][off], scale, zero), blockvec[k + 4], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 20) & 0xf][off], scale, zero), blockvec[k + 5], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 24) & 0xf][off], scale, zero), blockvec[k + 6], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 28) & 0xf][off], scale, zero), blockvec[k + 7], res2);
    i += width;
    k += 8;
    res += __low2float(res2) + __high2float(res2);
  }

  atomicAdd(&mul[b * width + w], res);
}

void vecquant3matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1) / 2;
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT3 - 1) / BLOCKHEIGHT3,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant3MatMulKernel<<<blocks, threads>>>(
    (half2*) vec.data_ptr(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    batch, vec_height, height, width
  );
}

__global__ void VecQuant3MatMulKernel(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
	  int batch,
	  int vec_height,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKHEIGHT3 * 16 / 3;
  int b = blockIdx.z;
  int h = BLOCKHEIGHT3 * blockIdx.x;
  int hr = (min(height, h + BLOCKHEIGHT3) - h) * 16 / 3;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];
  if (threadIdx.x < hr)
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * blockwidth2 + threadIdx.x];

  __shared__ half2 deq2[64][32];
  int val = threadIdx.x / 32;
  int off = threadIdx.x % 32;
  for (; val < 64; val += BLOCKWIDTH / 32) {
    deq2[val][off] = __halves2half2(
       __int2half_rn(val & 0x7), __int2half_rn(val >> 3)
    );
  }
  if (w >= width) return;

  int i = width * h + w;
  int k = 0;

  float res = 0;
  half2 res2;
  half2 scale = __float2half2_rn(scales[w] / 7.0f * 2.0f);
  half2 zero = __float2half2_rn(-scales[w]);

  unsigned int tmp1;
  unsigned int tmp2;
  unsigned int tmp;

  __syncthreads();

  while (k < hr) {
    std::memset(&res2, 0, sizeof(half2));
    tmp1 = as_unsigned(mat[i]);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
    i += width;
    tmp2 = as_unsigned(mat[i]);
    tmp = (tmp1 >> 30) | ((tmp2 << 2) & 0x3c);
    res2 = __hfma2(__hfma2(deq2[tmp][off], scale, zero), blockvec[k + 5], res2);
    tmp2 >>= 4;
    k += 6;
    res2 = __hfma2(__hfma2(deq2[(tmp2 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp2 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    i += width;
    tmp1 = as_unsigned(mat[i]);
    tmp = (tmp2 >> 24) | ((tmp1 << 4) & 0x30);
    res2 = __hfma2(__hfma2(deq2[tmp][off], scale, zero), blockvec[k + 4], res2);
    tmp1 >>= 2;
    k += 5;
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  0) & 0x3f][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >>  6) & 0x3f][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 12) & 0x3f][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 18) & 0x3f][off], scale, zero), blockvec[k + 3], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp1 >> 24) & 0x3f][off], scale, zero), blockvec[k + 4], res2);
    i += width;
    k += 5;
    res += __low2float(res2) + __high2float(res2);
  }
  atomicAdd(&mul[b * width + w], res);
}

void vecquant4matmul_cuda(
  torch::Tensor vec,
  torch::Tensor mat,
  torch::Tensor mul,
  torch::Tensor scales
) {
  int batch = vec.size(0);
  int vec_height = vec.size(1) / 2;
  int height = mat.size(0);
  int width = mat.size(1);

  dim3 blocks(
    (height + BLOCKHEIGHT4 - 1) / BLOCKHEIGHT4,
    (width + BLOCKWIDTH - 1) / BLOCKWIDTH,
    batch
  );
  dim3 threads(BLOCKWIDTH);

  VecQuant4MatMulKernel<<<blocks, threads>>>(
    (half2*) vec.data_ptr(),
    mat.data_ptr<int>(),
    mul.data_ptr<float>(),
    scales.data_ptr<float>(),
    batch, vec_height, height, width
  );
}

__global__ void VecQuant4MatMulKernel(
    const  half2* __restrict__ vec,
    const    int* __restrict__ mat,
           float* __restrict__ mul,
    const  float* __restrict__ scales,
    int batch,
	  int vec_height,
    int height,
    int width
) {
  const int blockwidth2 = BLOCKHEIGHT4 * 4;
  int b = blockIdx.z;
  int h = BLOCKHEIGHT4 * blockIdx.x;
  int hr = (min(height, h + BLOCKHEIGHT4) - h) * 4;
  int w = BLOCKWIDTH * blockIdx.y + threadIdx.x;

  __shared__ half2 blockvec[blockwidth2];
  if (threadIdx.x < hr)
    blockvec[threadIdx.x] = vec[b * vec_height + blockIdx.x * blockwidth2 + threadIdx.x];

  __shared__ half2 deq2[256][8];
  int val = threadIdx.x / 8;
  int off = threadIdx.x % 8;
  for (; val < 256; val += BLOCKWIDTH / 8) {
    deq2[val][off] = __halves2half2(
       __int2half_rn(val & 0xF), __int2half_rn(val >> 4)
    );
  }
  if (w >= width) return;

  int i = width * h + w;
  int k = 0;

  float res = 0;
  half2 res2;
  half2 scale = __float2half2_rn(scales[w] / 15.0f * 2.0f);
  half2 zero = __float2half2_rn(-scales[w]);

  unsigned int tmp;

  __syncthreads();

  while (k < hr) {
    std::memset(&res2, 0, sizeof(half2));
    tmp = as_unsigned(mat[i]);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  0) & 0xff][off], scale, zero), blockvec[k + 0], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >>  8) & 0xff][off], scale, zero), blockvec[k + 1], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 16) & 0xff][off], scale, zero), blockvec[k + 2], res2);
    res2 = __hfma2(__hfma2(deq2[(tmp >> 24) & 0xff][off], scale, zero), blockvec[k + 3], res2);
    i += width;
    k += 4;
    res += __low2float(res2) + __high2float(res2);
  }
  atomicAdd(&mul[b * width + w], res);
}
